#include "hip/hip_runtime.h"
#include "voxelGenerator.h"

using namespace nvinfer1;
using nvinfer1::VoxelGeneratorPlugin;
using nvinfer1::VoxelGeneratorPluginCreator;
using namespace std;

#define checkCudaErrors(status)                                   \
{                                                                 \
  if (status != 0)                                                \
  {                                                               \
    std::cout << "Cuda failure: " << hipGetErrorString(status)   \
              << " at line " << __LINE__                          \
              << " in file " << __FILE__                          \
              << " error status: " << status                      \
              << std::endl;                                       \
              abort();                                            \
    }                                                             \
}

#define CHECK(status) \
    do\
    {\
        auto ret = (status);\
        if (ret != 0)\
        {\
            std::cerr << "Cuda failure: " << ret << std::endl;\
            abort();\
        }\
    } while (0)



#define CUDA_MEM_ALIGN 256

static const char* PLUGIN_VERSION{"1"};
static const char* PLUGIN_NAME{"VoxelGeneratorPlugin"};

// Static class fields initialization
PluginFieldCollection VoxelGeneratorPluginCreator::mFC{};
std::vector<PluginField> VoxelGeneratorPluginCreator::mPluginAttributes;

// Helper function for serializing plugin
template <typename T>
void writeToBuffer(char*& buffer, const T& val)
{
    *reinterpret_cast<T*>(buffer) = val;
    buffer += sizeof(T);
}

// Helper function for deserializing plugin
template <typename T>
T readFromBuffer(const char*& buffer)
{
    T val = *reinterpret_cast<const T*>(buffer);
    buffer += sizeof(T);
    return val;
}

// Mimic np.round as in voxel generator in spconv implementation
int np_round(float x) {
  // half way round to nearest-even
  int x2 = int(x * 2.0f);
  if(x != int(x) && x2 == x * 2.0f) {
    return int(x / 2.0f + 0.5f) * 2;
  }
  return int(x + 0.5f);
}

// ALIGNPTR
int8_t* alignPtr(int8_t* ptr, uintptr_t to)
{
    uintptr_t addr = (uintptr_t) ptr;
    if (addr % to)
    {
        addr += to - addr % to;
    }
    return (int8_t*) addr;
}

// NEXTWORKSPACEPTR
int8_t* nextWorkspacePtr(int8_t* ptr, uintptr_t previousWorkspaceSize)
{
    uintptr_t addr = (uintptr_t) ptr;
    addr += previousWorkspaceSize;
    return alignPtr((int8_t*) addr, CUDA_MEM_ALIGN);
}

// CALCULATE TOTAL WORKSPACE SIZE
size_t calculateTotalWorkspaceSize(size_t* workspaces, int count)
{
    size_t total = 0;
    for (int i = 0; i < count; i++)
    {
        total += workspaces[i];
        if (workspaces[i] % CUDA_MEM_ALIGN)
        {
            total += CUDA_MEM_ALIGN - (workspaces[i] % CUDA_MEM_ALIGN);
        }
    }
    return total;
}

// create the plugin at runtime from a byte stream
VoxelGeneratorPlugin::VoxelGeneratorPlugin(
int max_voxels, int max_points, int voxel_features, float x_min,
float x_max, float y_min, float y_max, float z_min, float z_max,
float voxel_x, float voxel_y, float voxel_z
) : voxelNum_(max_voxels), pointNum_(max_points), featureNum_(voxel_features),
    min_x_range_(x_min), max_x_range_(x_max), min_y_range_(y_min),
    max_y_range_(y_max), min_z_range_(z_min), max_z_range_(z_max),
    voxel_x_size_(voxel_x), voxel_y_size_(voxel_y),
    voxel_z_size_(voxel_z)
{
}

VoxelGeneratorPlugin::VoxelGeneratorPlugin(
    int max_voxels, int max_points, int voxel_features, float x_min,
    float x_max, float y_min, float y_max, float z_min, float z_max,
    float voxel_x, float voxel_y, float voxel_z, int point_features,
    int grid_x, int grid_y, int grid_z
) : voxelNum_(max_voxels), pointNum_(max_points), featureNum_(voxel_features),
    min_x_range_(x_min), max_x_range_(x_max), min_y_range_(y_min),
    max_y_range_(y_max), min_z_range_(z_min), max_z_range_(z_max),
    voxel_x_size_(voxel_x), voxel_y_size_(voxel_y),
    voxel_z_size_(voxel_z), pointFeatureNum_(point_features),
    grid_x_size_(grid_x), grid_y_size_(grid_y), grid_z_size_(grid_z)
{
}

VoxelGeneratorPlugin::VoxelGeneratorPlugin(const void* data, size_t length)
{
    const char* d = reinterpret_cast<const char*>(data);
    voxelNum_ = readFromBuffer<int>(d);
    pointNum_ = readFromBuffer<int>(d);
    featureNum_ = readFromBuffer<int>(d);
    min_x_range_ = readFromBuffer<float>(d);
    max_x_range_ = readFromBuffer<float>(d);
    min_y_range_ = readFromBuffer<float>(d);
    max_y_range_ = readFromBuffer<float>(d);
    min_z_range_ = readFromBuffer<float>(d);
    max_z_range_ = readFromBuffer<float>(d);
    voxel_x_size_ = readFromBuffer<float>(d);
    voxel_y_size_ = readFromBuffer<float>(d);
    voxel_z_size_ = readFromBuffer<float>(d);
    pointFeatureNum_ = readFromBuffer<int>(d);
    grid_x_size_ = readFromBuffer<int>(d);
    grid_y_size_ = readFromBuffer<int>(d);
    grid_z_size_ = readFromBuffer<int>(d);
}

IPluginV2DynamicExt* VoxelGeneratorPlugin::clone() const noexcept
{
    auto* plugin = new VoxelGeneratorPlugin(voxelNum_, pointNum_, featureNum_, min_x_range_, max_x_range_,
        min_y_range_, max_y_range_, min_z_range_, max_z_range_, voxel_x_size_, voxel_y_size_, voxel_z_size_,
        pointFeatureNum_, grid_x_size_, grid_y_size_, grid_z_size_);
    plugin->setPluginNamespace(mNamespace.c_str());
    return plugin;
}

nvinfer1::DimsExprs VoxelGeneratorPlugin::getOutputDimensions(
int outputIndex, const nvinfer1::DimsExprs* inputs, int nbInputs, nvinfer1::IExprBuilder& exprBuilder) noexcept
{
    // assert(outputIndex >= 0 && outputIndex < this->getNbOutputs());
    auto batch_size = inputs[0].d[0];
    // auto batch_size = 1;
    // std::cout  << inputs[0].nbDims << " " << inputs[0].d[0]->getConstantValue() << " " << inputs[0].d[1]->getConstantValue() << " " << inputs[0].d[2]->getConstantValue() << std::endl;
    // std::cout  << inputs[1].nbDims << " " << inputs[1].d[0]->getConstantValue() << std::endl;
    if (outputIndex == 0)
    {
        // std::cout << "batch_size: " << batch_size->getConstantValue() << " voxel_num: " << voxelNum_ << " featurennum_: " << featureNum_ << std::endl;
        nvinfer1::DimsExprs dim0{};
        dim0.nbDims = 3;
        dim0.d[0] = batch_size;
        dim0.d[1] = exprBuilder.constant(voxelNum_);
        dim0.d[2] = exprBuilder.constant(featureNum_);
        return dim0; // voxels 1 20000 4
    }
    if(outputIndex == 1){
        // std::cout << "batch_size: " << batch_size->getConstantValue() << "  voxel_num: " << voxelNum_ << " featurennum_: " << 4 << std::endl;
        nvinfer1::DimsExprs dim1{};
        dim1.nbDims = 3;
        dim1.d[0] = batch_size;
        dim1.d[1] = exprBuilder.constant(voxelNum_);
        dim1.d[2] = exprBuilder.constant(4);
        return dim1; // coors 1 20000 4
    }
    if(outputIndex == 2)
    {   
        // std::cout << "batch_size: " << batch_size->getConstantValue() << std::endl;
        nvinfer1::DimsExprs dim2{};
        dim2.nbDims = 1;
        dim2.d[0] = batch_size;
        return dim2;
    }
}

bool VoxelGeneratorPlugin::supportsFormatCombination(
int pos, const nvinfer1::PluginTensorDesc* inOut, int nbInputs, int nbOutputs) noexcept
{
    // PLUGIN_ASSERT(nbInputs == 2);
    // PLUGIN_ASSERT(nbOutputs == 2);
    const PluginTensorDesc& in = inOut[pos];
    if (pos == 0)       // PointCloud Array --- x, y, z, i   dim: 1  40000 4
    {
        return (in.type == nvinfer1::DataType::kFLOAT) && (in.format == TensorFormat::kLINEAR);
    }
    if (pos == 1)       // Point Num
    {
        return (in.type == nvinfer1::DataType::kINT32) && (in.format == TensorFormat::kLINEAR);
    }
    if (pos == 2)       // voxels, dim: 1 20000 4
    {
        return (in.type == nvinfer1::DataType::kFLOAT) && (in.format == TensorFormat::kLINEAR);
    }
    if (pos == 3)       // voxelCoords, dim: 1 x 20000 x 4
    {
        return (in.type == nvinfer1::DataType::kINT32) && (in.format == TensorFormat::kLINEAR);
    }
    if (pos == 4)    // voxel_num valid
    {
        return (in.type == nvinfer1::DataType::kINT32) && (in.format == TensorFormat::kLINEAR);
    }
    
    return false;
}

void VoxelGeneratorPlugin::configurePlugin(const nvinfer1::DynamicPluginTensorDesc* in, int nbInputs,
const nvinfer1::DynamicPluginTensorDesc* out, int nbOutputs) noexcept
{
    pointFeatureNum_ = in[0].desc.dims.d[2];
    grid_x_size_ = np_round((max_x_range_ - min_x_range_) / voxel_x_size_);
    grid_y_size_ = np_round((max_y_range_ - min_y_range_) / voxel_y_size_);
    grid_z_size_ = np_round((max_z_range_ - min_z_range_) / voxel_z_size_);
}

size_t VoxelGeneratorPlugin::getWorkspaceSize(const nvinfer1::PluginTensorDesc* inputs, int nbInputs,
    const nvinfer1::PluginTensorDesc* outputs, int nbOutputs) const noexcept
{
    int batchSize = inputs[0].dims.d[0];
    size_t coor_to_voxelidx_size = batchSize * grid_z_size_ * grid_y_size_ * grid_x_size_ * 2 * sizeof(unsigned int);
    size_t num_points_per_voxel_size = batchSize * voxelNum_ * 1 * sizeof(unsigned int);
    // size_t grid_voxel_size = batchSize * grid_z_size_ * grid_y_size_ * grid_x_size_ * pointNum_ * pointFeatureNum_ * sizeof(float);
    size_t voxel_size = batchSize * voxelNum_ * pointNum_ * featureNum_ * sizeof(float);

    size_t workspaces[3];
    workspaces[0] = coor_to_voxelidx_size;
    workspaces[1] = num_points_per_voxel_size;
    workspaces[2] = voxel_size;
   
    return  calculateTotalWorkspaceSize(workspaces, 3);
}

// 1 N 5 4 ---> 1 N 4
__global__ void generateAverage_kernel(float *voxel_,
        float *voxel_features_data,unsigned int* num_points_per_voxel,unsigned int *voxel_num_data)
{
    int voxelidx = blockIdx.x * blockDim.x + threadIdx.x;
    if(voxelidx >= *voxel_num_data) return;

    float4 point = ((float4*)voxel_)[voxelidx*MAX_POINTS_PER_VOXEL];
    // float4 point = ((float4*)points)[point_idx];
    int index_z = floorf((point.z - (-3.0)) / 0.1);
    int index_y = floorf((point.y - (-40.)) / 0.05);
    int index_x = floorf((point.x - 0.0)/0.05);

    float sum_x = 0.0;
    float sum_y = 0.0;
    float sum_z = 0.0;
    float sum_i = 0.0;

    int num_point = num_points_per_voxel[voxelidx];

    sum_x = (*(voxel_+ voxelidx*MAX_POINTS_PER_VOXEL*4+0*4) + 
            *(voxel_+ voxelidx*MAX_POINTS_PER_VOXEL*4+1*4) + 
            *(voxel_+ voxelidx*MAX_POINTS_PER_VOXEL*4+2*4) +
            *(voxel_+ voxelidx*MAX_POINTS_PER_VOXEL*4+3*4) + 
            *(voxel_+ voxelidx*MAX_POINTS_PER_VOXEL*4+4*4))/num_point;

    sum_y = (*(voxel_+ voxelidx*MAX_POINTS_PER_VOXEL*4+0*4 + 1) + 
            *(voxel_+ voxelidx*MAX_POINTS_PER_VOXEL*4+1*4 + 1) +
            *(voxel_+ voxelidx*MAX_POINTS_PER_VOXEL*4+2*4 + 1) + 
            *(voxel_+ voxelidx*MAX_POINTS_PER_VOXEL*4+3*4 + 1) + 
            *(voxel_+ voxelidx*MAX_POINTS_PER_VOXEL*4+4*4 + 1))/num_point;

    sum_z = (*(voxel_+ voxelidx*MAX_POINTS_PER_VOXEL*4+0*4 + 2) + 
            *(voxel_+ voxelidx*MAX_POINTS_PER_VOXEL*4+1*4 + 2) + 
            *(voxel_+ voxelidx*MAX_POINTS_PER_VOXEL*4+2*4 + 2) + 
            *(voxel_+ voxelidx*MAX_POINTS_PER_VOXEL*4+3*4 + 2) +
            *(voxel_+ voxelidx*MAX_POINTS_PER_VOXEL*4+4*4 + 2))/num_point;

    sum_i = (*(voxel_+ voxelidx*MAX_POINTS_PER_VOXEL*4+0*4 + 3) +
            *(voxel_+ voxelidx*MAX_POINTS_PER_VOXEL*4+1*4 + 3) +
            *(voxel_+ voxelidx*MAX_POINTS_PER_VOXEL*4+2*4 + 3) +
            *(voxel_+ voxelidx*MAX_POINTS_PER_VOXEL*4+3*4 + 3) +
            *(voxel_+ voxelidx*MAX_POINTS_PER_VOXEL*4+4*4 + 3))/num_point;
    
    #if 0
     if (index_z == 32 && index_y == 731 && index_x == 95)
     {
         printf("sum: %f,%f,%f,%f\n",sum_x,sum_y,sum_z,sum_i);
     }
     #endif

    // float x = sum_x/num_point;
    // float y = sum_y/num_point;
    // float z = sum_z/num_point;
    // float inten = sum_i / num_point;

    float *address = voxel_features_data + voxelidx*4;
    atomicExch(address+0,sum_x);
    atomicExch(address+1,sum_y);
    atomicExch(address+2,sum_z);
    atomicExch(address+3,sum_i);
    #if 0
    if (index_z == 32 && index_y == 731 && index_x == 95)
     {
         printf("address_sum: %f,%f,%f,%f\n",*(address),*(address+1),*(address+2),*(address+3));
     }
    #endif
}


hipError_t generateAverage_launch(float *voxel_,
        float *voxel_features_data,unsigned int* num_points_per_voxel,unsigned int* voxel_num_data,
        hipStream_t stream)
{
  int threadNum = THREADS_FOR_VOXEL;
  dim3 blocks((MAX_VOXELS+threadNum-1)/threadNum);
  dim3 threads(threadNum);
  generateAverage_kernel<<<blocks, threads, 0, stream>>>
       (voxel_,voxel_features_data,num_points_per_voxel,voxel_num_data);
  hipError_t err = hipGetLastError();
  return err;
}

__device__ void cuda_sleep(int64_t num_cycles)
{
    int64_t cycles = 0;
    int64_t start = clock64();
    while(cycles < num_cycles)
    {
        cycles = clock64() - start;
    }
}


__global__ void generateVoxels_kernel(float *points, unsigned int* points_size,
        float min_x_range, float max_x_range,
        float min_y_range, float max_y_range,
        float min_z_range, float max_z_range,
        float voxel_x_size, float voxel_y_size, float voxel_z_size,
        int grid_x_size, int grid_y_size,int grid_z_size,
        unsigned int *coor_to_voxelidx,unsigned int *coords_data, float* voxel_,
        unsigned int *num_points_per_voxel,unsigned int *voxel_num_data)
{
    // printf("point_size:%d\n",*points_size);
    int point_idx = blockIdx.x * blockDim.x + threadIdx.x;
    if(point_idx >= *points_size) return;
    // printf("generatevoxel11111111");
    float4 point = ((float4*)points)[point_idx];
    
    if( !(point.x >= min_x_range && point.x < max_x_range
        && point.y >= min_y_range && point.y < max_y_range
        && point.z >= min_z_range && point.z < max_z_range) ) {
      return;
    }
    // printf("generatevoxel222222\n");
    int index_x = floorf((point.x - min_x_range) / voxel_x_size);
    int index_y = floorf((point.y - min_y_range) / voxel_y_size);
    int index_z = floorf((point.z - min_z_range) / voxel_z_size);
    // printf("index_x: %d,%f,%f,%f\n",index_x,point.x,min_x_range,voxel_x_size);
    // printf("index_i: %d,%d,%d\n",index_x,index_y,index_z);
    // bool failed = false;
    if ((index_x < 0) or (index_x >= grid_x_size) or (index_y < 0) or (index_y >= grid_y_size) or (index_z < 0) or (index_z >= grid_z_size))
        return;
    
    unsigned int voxel_index = index_z * (grid_y_size * grid_x_size) + index_y * grid_x_size + index_x;
                                // index_z * (grid_y_size * grid_x_size * 2) + index_y*grid_z_size *2 + index_x*2 + 
    unsigned int point_id = atomicAdd(coor_to_voxelidx+voxel_index*2,1);


    if(point_id >= MAX_POINTS_PER_VOXEL) return;

    unsigned int current_voxelid = 0;

    if (point_id == 0)
    {
        //保存coor and current_voxel_id
        current_voxelid = atomicAdd(voxel_num_data,1);
        #if 0
        if(current_voxelid == 0)
        {
            printf("current_voxelid:%d,%f,%f,%f,%d,%d,%d,%d\n",current_voxelid,point.x,point.y,point.z,index_x,index_y,index_z,voxel_index * 2 + 1);
        }
        #endif

        //save current_voxelid
        unsigned int *current_voxelid_address = coor_to_voxelidx + voxel_index * 2 + 1;
        atomicExch(current_voxelid_address,current_voxelid);


        //save coord
        uint4 coord = {0,index_z,index_y,index_x};
        ((uint4*)coords_data)[current_voxelid] = coord;

    }
    // nanosleep()
    else{
        // if(current_voxelid == 0 && )
        // __nanosleep(100);
        current_voxelid = coor_to_voxelidx[voxel_index*2+1];
        if(current_voxelid == 0)
        {
            cuda_sleep(300000); //10000000
            current_voxelid = coor_to_voxelidx[voxel_index*2+1];
        }
        #if 0
         if(current_voxelid == 0)
        {
            printf("current_voxelid:%d,%f,%f,%f,%d,%d,%d,%d,%d\n",current_voxelid,point.x,point.y,point.z,index_x,index_y,index_z,point_id,voxel_index*2+1);
        }
        #endif
    }
    //save point
    float *address = voxel_ + current_voxelid*MAX_POINTS_PER_VOXEL*4+point_id*4;
    atomicExch(address+0,point.x);
    atomicExch(address+1,point.y);
    atomicExch(address+2,point.z);
    atomicExch(address+3,point.w);

     //num_points_per_voxel ++
    unsigned int *num_points_per_voxel_address = num_points_per_voxel + current_voxelid;
    unsigned int num = *(coor_to_voxelidx+voxel_index*2);
    if(num > MAX_POINTS_PER_VOXEL)
        num = MAX_POINTS_PER_VOXEL;
    atomicExch(num_points_per_voxel_address,num);
    // [  0  27 843  72]
    #if 0
    if (index_z == 32 && index_y == 731 && index_x == 95)
    {
        printf("point: %f,%f,%f,%f %d\n",point.x,point.y,point.z,point.w,point_id);
        printf("adress: %f,%f,%f,%f\n",*address,*(address+1),*(address+2),*(address+3));
        printf("fsfsfsfsf current_voxelid: %d\n",current_voxelid);
        printf("coor_to_voxelidx[voxel_index*2]: %d\n",atomicAdd(coor_to_voxelidx+voxel_index*2,0));
        printf("point_id: %d\n",point_id);
        printf("num_points_per_voxel_address: %d\n",*num_points_per_voxel_address);
    }
    #endif

}

hipError_t generateVoxels_launch(float *points, unsigned int* points_size,
        float min_x_range, float max_x_range,
        float min_y_range, float max_y_range,
        float min_z_range, float max_z_range,
        float voxel_x_size, float voxel_y_size, float voxel_z_size,
        int grid_x_size, int grid_y_size,int grid_z_size,
        unsigned int *coor_to_voxelidx,unsigned int *coords_data, float *voxel_,
        unsigned int *num_points_per_voxel,unsigned int * voxel_num_data, 
        hipStream_t stream)
{
  int threadNum = THREADS_FOR_VOXEL;

  dim3 blocks((MAX_POINTS+threadNum-1)/threadNum);
  dim3 threads(threadNum);
  generateVoxels_kernel<<<blocks, threads, 0, stream>>>
       (points, points_size,
        min_x_range, max_x_range,
        min_y_range, max_y_range,
        min_z_range, max_z_range,
        voxel_x_size, voxel_y_size, voxel_z_size,
        grid_x_size, grid_y_size,grid_z_size,
        coor_to_voxelidx,coords_data,voxel_,num_points_per_voxel,voxel_num_data);
  hipError_t err = hipGetLastError();
  return err;
}


int VoxelGeneratorPlugin::enqueue(const nvinfer1::PluginTensorDesc* inputDesc,
    const nvinfer1::PluginTensorDesc* outputDesc, const void* const* inputs, void* const* outputs, void* workspace,
    hipStream_t stream) noexcept
{
    int batchSize = inputDesc[0].dims.d[0];
    // int maxNumPoints = inputDesc[0].dims.d[1];
    //TRT-input
    // std::cout << "voxelgenerator batch_size: " << batchSize << std::endl;
    float * pointCloud = const_cast<float *>((const float *)inputs[0]);
    unsigned int* pointNum = const_cast<unsigned int *>((const unsigned int *)inputs[1]);
    //TRT-output
    float *voxel_features_data = (float *)(outputs[0]);
    unsigned int *coords_data = (unsigned int *)(outputs[1]);
    unsigned int *voxel_num_data = (unsigned int *)(outputs[2]);
    // unsigned int *params_data = (unsigned int *)(outputs[2]);
    int dense_voxel_num = grid_z_size_ * grid_y_size_ * grid_x_size_;
    size_t coor_to_voxelidx_size = batchSize * dense_voxel_num * 2* sizeof(unsigned int);
    size_t num_points_per_voxel_size = batchSize * voxelNum_ * 1 * sizeof(unsigned int);
    // size_t grid_voxel_size = batchSize * grid_z_size_ * grid_y_size_ * grid_x_size_ * pointNum_ * featureNum_ * sizeof(float);
    size_t voxel_size = batchSize * voxelNum_ * pointNum_ * featureNum_ * sizeof(float);
    size_t workspaces[3];
    workspaces[0] = coor_to_voxelidx_size;
    workspaces[1] = num_points_per_voxel_size;
    workspaces[2] = voxel_size;
    size_t total_workspace = calculateTotalWorkspaceSize(workspaces, 3);
    // std::cout << "enqueue11111" << std::endl;
    unsigned int* coor_to_voxelidx = static_cast<unsigned int*>(workspace);
    unsigned int* num_points_per_voxel = reinterpret_cast<unsigned int*>(
        nextWorkspacePtr(reinterpret_cast<int8_t*>(coor_to_voxelidx), coor_to_voxelidx_size)
    );
    // float* grid_voxel_ = reinterpret_cast<float*>(
    //     nextWorkspacePtr(reinterpret_cast<int8_t*>(num_points_per_voxel), num_points_per_voxel_size)
    // );
    float* voxel_ = reinterpret_cast<float*>(
        nextWorkspacePtr(reinterpret_cast<int8_t*>(num_points_per_voxel), num_points_per_voxel_size)
    );
    // Initialize workspace memory
    checkCudaErrors(hipMemsetAsync(coor_to_voxelidx, 0, total_workspace, stream)); // total_workspace
    // checkCudaErrors(hipMemsetAsync(num_points_per_voxel,0,num_points_per_voxel_size,stream));
    // checkCudaErrors(hipMemsetAsync(voxel_,0,voxel_size,stream));
    unsigned int voxel_features_data_size = batchSize * voxelNum_  * featureNum_ * sizeof(float);
    unsigned int coords_data_size = batchSize * voxelNum_ * 4 * sizeof(unsigned int);
    unsigned int voxel_num_data_size = batchSize * sizeof(unsigned int);
    // unsigned int params_data_size = batchSize * sizeof(unsigned int);
    checkCudaErrors(hipMemsetAsync(voxel_features_data, 0, voxel_features_data_size, stream));
    checkCudaErrors(hipMemsetAsync(coords_data, 0, coords_data_size, stream));
    checkCudaErrors(hipMemsetAsync(voxel_num_data, 0, voxel_num_data_size, stream));
    // checkCudaErrors(hipMemsetAsync(params_data, 0, params_data_size, stream));

    checkCudaErrors(generateVoxels_launch(
          pointCloud, pointNum,
          min_x_range_, max_x_range_,
          min_y_range_, max_y_range_,
          min_z_range_, max_z_range_,
          voxel_x_size_, voxel_y_size_, voxel_z_size_,
          grid_x_size_, grid_y_size_, grid_z_size_,
          coor_to_voxelidx,coords_data, voxel_,num_points_per_voxel,voxel_num_data, stream));

    
    checkCudaErrors(generateAverage_launch(
        voxel_,voxel_features_data,num_points_per_voxel,voxel_num_data, stream));
    return 0;
}

nvinfer1::DataType VoxelGeneratorPlugin::getOutputDataType(
    int index, const nvinfer1::DataType* inputTypes, int nbInputs) const noexcept
{
    if(index == 0)
      return inputTypes[0];
    return inputTypes[1];
}

const char* VoxelGeneratorPlugin::getPluginType() const noexcept
{
    return PLUGIN_NAME;
}

const char* VoxelGeneratorPlugin::getPluginVersion() const noexcept
{
    return PLUGIN_VERSION;
}

int VoxelGeneratorPlugin::getNbOutputs() const noexcept
{
    return 3;
}

int VoxelGeneratorPlugin::initialize() noexcept
{
    return 0;
}

void VoxelGeneratorPlugin::terminate() noexcept
{
}

size_t VoxelGeneratorPlugin::getSerializationSize() const noexcept
{
    return 9 * sizeof(float) + 7 * sizeof(int);
}

void VoxelGeneratorPlugin::serialize(void* buffer) const noexcept
{
    char* d = reinterpret_cast<char*>(buffer);
    writeToBuffer<int>(d, voxelNum_);
    writeToBuffer<int>(d, pointNum_);
    writeToBuffer<int>(d, featureNum_);
    writeToBuffer<float>(d, min_x_range_);
    writeToBuffer<float>(d, max_x_range_);
    writeToBuffer<float>(d, min_y_range_);
    writeToBuffer<float>(d, max_y_range_);
    writeToBuffer<float>(d, min_z_range_);
    writeToBuffer<float>(d, max_z_range_);
    writeToBuffer<float>(d, voxel_x_size_);
    writeToBuffer<float>(d, voxel_y_size_);
    writeToBuffer<float>(d, voxel_z_size_);
    writeToBuffer<int>(d, pointFeatureNum_);
    writeToBuffer<int>(d, grid_x_size_);
    writeToBuffer<int>(d, grid_y_size_);
    writeToBuffer<int>(d, grid_z_size_);
}

void VoxelGeneratorPlugin::destroy() noexcept
{
    delete this;
}

void VoxelGeneratorPlugin::setPluginNamespace(const char* libNamespace) noexcept
{
    mNamespace = libNamespace;
}

const char* VoxelGeneratorPlugin::getPluginNamespace() const noexcept
{
    return mNamespace.c_str();
}


VoxelGeneratorPluginCreator::VoxelGeneratorPluginCreator()
{
    
    mPluginAttributes.clear();

    // std::cout <<  *max_num_points_per_voxel_ptr << std::endl;
    mPluginAttributes.emplace_back(PluginField("max_num_points_per_voxel", max_num_points_per_voxel_ptr, PluginFieldType::kINT32, 1));
    mPluginAttributes.emplace_back(PluginField("max_voxels", max_voxels_ptr, PluginFieldType::kINT32, 1));
    mPluginAttributes.emplace_back(PluginField("point_cloud_range", point_cloud_range_ptr, PluginFieldType::kFLOAT32, 1));
    mPluginAttributes.emplace_back(PluginField("voxel_feature_num", voxel_feature_num_ptr, PluginFieldType::kINT32, 1));
    mPluginAttributes.emplace_back(PluginField("voxel_size", voxel_size_ptr, PluginFieldType::kFLOAT32, 1));

    mFC.nbFields = mPluginAttributes.size();
    mFC.fields = mPluginAttributes.data();
}

const char* VoxelGeneratorPluginCreator::getPluginName() const noexcept
{
    return PLUGIN_NAME;
}

const char* VoxelGeneratorPluginCreator::getPluginVersion() const noexcept
{
    return PLUGIN_VERSION;
}

const PluginFieldCollection* VoxelGeneratorPluginCreator::getFieldNames() noexcept
{
    return &mFC;
}

IPluginV2* VoxelGeneratorPluginCreator::createPlugin(const char* name, const PluginFieldCollection* fc) noexcept
{
    const PluginField* fields = fc->fields;
    int nbFields = fc->nbFields;
    int max_points = 0;
    int max_voxels = 0;
    int voxel_feature_num = 0;
    float point_cloud_range[6] = {0.0f};
    float voxel_size[3] = {0.0f};
    for (int i = 0; i < nbFields; ++i)
    {
        const char* attr_name = fields[i].name;
        if (!strcmp(attr_name, "max_num_points_per_voxel"))
        {
            const int* d = static_cast<const int*>(fields[i].data);
            max_points = d[0];
        }
        else if (!strcmp(attr_name, "max_voxels"))
        {
            const int* d = static_cast<const int*>(fields[i].data);
            max_voxels = d[0];
        }
        else if (!strcmp(attr_name, "point_cloud_range"))
        {
            const float* d = static_cast<const float*>(fields[i].data);
            point_cloud_range[0] = d[0];
            point_cloud_range[1] = d[1];
            point_cloud_range[2] = d[2];
            point_cloud_range[3] = d[3];
            point_cloud_range[4] = d[4];
            point_cloud_range[5] = d[5];
        }
        else if (!strcmp(attr_name, "voxel_feature_num"))
        {
            const int* d = static_cast<const int*>(fields[i].data);
            voxel_feature_num = d[0];
        }
        else if (!strcmp(attr_name, "voxel_size"))
        {
            const float* d = static_cast<const float*>(fields[i].data);
            voxel_size[0] = d[0];
            voxel_size[1] = d[1];
            voxel_size[2] = d[2];
        }
    }
    // std::cout << max_voxels << " " << max_points << " " <<voxel_feature_num << " " << point_cloud_range[0] << " " << point_cloud_range[1] << " "
    // << point_cloud_range[2] << " "<< point_cloud_range[3] << " " << point_cloud_range[4] << " " << point_cloud_range[5] << " " << voxel_size[0] << " "
    // << voxel_size[1] << " " << voxel_size[2] << std::endl;
    IPluginV2DynamicExt* plugin = new VoxelGeneratorPlugin(max_voxels, max_points,voxel_feature_num, point_cloud_range[0],
        point_cloud_range[3], point_cloud_range[1], point_cloud_range[4], point_cloud_range[2],
        point_cloud_range[5], voxel_size[0], voxel_size[1], voxel_size[2]);
    return plugin;
}

IPluginV2* VoxelGeneratorPluginCreator::deserializePlugin(
    const char* name, const void* serialData, size_t serialLength) noexcept
{
    return new VoxelGeneratorPlugin(serialData, serialLength);
}

void VoxelGeneratorPluginCreator::setPluginNamespace(const char* libNamespace) noexcept
{
    mNamespace = libNamespace;
}

const char* VoxelGeneratorPluginCreator::getPluginNamespace() const noexcept
{
    return mNamespace.c_str();
}

VoxelGeneratorPluginCreator::~VoxelGeneratorPluginCreator()
{
    delete max_num_points_per_voxel_ptr;
    delete max_voxels_ptr;
    delete [] voxel_size_ptr;
    delete [] point_cloud_range_ptr;
    delete voxel_feature_num_ptr;
}